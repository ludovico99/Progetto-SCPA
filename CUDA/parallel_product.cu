#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <math.h>

#include <hip/hip_runtime.h> // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h> // For CUDA SDK timers
#include "../header.h"

__global__ void CSR_kernel(const int M, const int K, const int nz, double *d_as, int *d_ja, int *d_irp, double *d_X, double *d_y, int numElements)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    int i = tid / K;
    int z = tid % K;
    double partial_sum = 0;
    if (tid < numElements)
    {
        if (i == 0 && d_irp[i] == -1)
        {
            d_y[i * K + z] = 0.0;
        }
        if (i > 0 && d_irp[i] == d_irp[i - 1])
        {
            d_y[i * K + z] = 0.0;
        }
        else
        {
            for (int j = d_irp[i]; (i < (M - 1) && j < d_irp[i + 1]) || (i >= M - 1 && j < nz); j++)
            {
                if (d_as != NULL)
                    partial_sum += d_as[j] * d_X[d_ja[j] * K + z];
                else
                    partial_sum += 1.0 * d_X[d_ja[j] * K + z];
            }
            d_y[i * K + z] = partial_sum;
        }
    }
}

double *CSR_GPU(int M, int N, int K, int nz, double *h_as, int *h_ja, int *h_irp, double **X, double *time)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    hipEvent_t start, stop;
    hipStream_t stream = NULL;

    double *h_y = NULL;
    double *h_X = NULL;
    double *d_y = NULL;

    double *d_X = NULL;

    double *d_as = NULL;
    int *d_ja = NULL;
    int *d_irp = NULL;

    float expireTimeMsec = 0.0;

    h_X = convert_2D_to_1D(M, K, X);

    h_y = (double *)malloc(M * K * sizeof(double));
    if (h_y == NULL)
    {
        printf("Errore malloc per y\n");
        exit(1);
    }

    // for (int i = 0; i < M; i++)
    // {
    //     h_y[i] = (double *)malloc(K * sizeof(double));
    //     if (h_y[i] == NULL)
    //     {
    //         printf("Errore malloc\n");
    //         exit(1);
    //     }
    // }
    printf("Allocating device variables for CPU CSR product ...\n");

    err = hipMalloc((void **)&d_y, M * K * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device y (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_X, N * K * sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device irp (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_as, nz * sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device as (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_ja, nz * sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device ja (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_irp, M * sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device irp (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // err = hipMemset(d_as, 0.0, sizeof(double) * nz);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset as (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // err = hipMemset(d_irp, -1, sizeof(int) * M);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset irp (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // err = hipMemset(d_ja, 0, sizeof(int) * nz);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset ja (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // err = hipMemset2D(d_X, X_pitch, 0, X_width, X_height);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset ja (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // err = hipMemset2D(d_y, y_pitch, 0, y_width, y_height);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset ja (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");

    err = hipMemcpy(d_as, h_as, nz * sizeof(double), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy as from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_ja, h_ja, nz * sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy ja from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_irp, h_irp, M * sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy irp from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_X, h_X, N * K * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy matrix X from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int numElements = M * K;
    int threadsPerBlock = 1024;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
           threadsPerBlock);

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // START TIMER
    checkCudaErrors(hipEventRecord(start, stream));

    CSR_kernel<<<blocksPerGrid, threadsPerBlock>>>(M, K, nz, d_as, d_ja, d_irp, d_X, d_y, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch CSR kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // STOP TIMER
    checkCudaErrors(hipEventRecord(stop, stream));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&expireTimeMsec, start, stop));

    printf("ELAPSED TIME FOR PARALLEL PRODUCT GPU: %lf ns = %lf ms = %lf seconds\n", expireTimeMsec * 1e6, expireTimeMsec, expireTimeMsec * 1e-3);

    if (time != NULL)
        *time = expireTimeMsec * 1e6;
    printf("GFLOPS FOR PARALLEL PRODUCT GPU: %lf\n", compute_GFLOPS(K, nz, expireTimeMsec * 1e6));

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_y, d_y, M * K * sizeof(double), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_as);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device as(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_ja);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device ja(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_irp);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device irp(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_X);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix X (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_y);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix y (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    printf("Freeing host memory ...\n");
    free(h_X);

    printf("Completed parallel product ...\n");

    // for (int i = 0; i < M; i++)
    // {
    //     printf("\n");
    //     for (int z = 0; z < K; z++)
    //     {
    //         printf("y[%d][%d] = %.70lf\t", i, z, h_y[i*K + z]);
    //     }
    //     printf("\n");
    // }

    return h_y;
}

__global__ void ELLPACK_kernel(const int M, const int K, int *nz_per_row, int * sum_nz, double *d_values, int *d_col_indices, double *d_X, double *d_y, int numElements)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    int i = tid / K;
    int z = tid % K;
    double partial_sum = 0.0;
    int offset = sum_nz[i];
    if (tid < numElements)
    {
        if (nz_per_row[i] == 0)
            d_y[i * K + z] = 0.0;
        else
        {
            for (int j = 0; j < nz_per_row[i]; j++)
            {
                if (d_values != NULL)
                    partial_sum += d_values[i * offset + j] * d_X[d_col_indices[i * offset + j] * K + z];
                else
                    partial_sum += 1.0 * d_X[d_col_indices[i * offset + j] * K + z];
            }
            d_y[i * K + z] = partial_sum;
        }
    }
}

double *ELLPACK_GPU(int M, int N, int K, int nz, int *nz_per_row, double **values, int **col_indices, double **X, double *time)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    hipEvent_t start, stop;
    hipStream_t stream = NULL;

    double *h_y = NULL;
    double *d_y = NULL;

    double *h_X = NULL;
    double *d_X = NULL;

    double *h_values = NULL;
    int *h_col_indices = NULL;

    double *d_values = NULL;
    int *d_col_indices = NULL;
    int *d_nz_per_row = NULL;

    int * h_sum_nz = NULL;
    int * d_sum_nz = NULL;

    float expireTimeMsec = 0.0;

    h_X = convert_2D_to_1D(M, K, X);

    h_y = (double *)malloc(M * K * sizeof(double));
    if (h_y == NULL)
    {
        printf("Errore malloc per y\n");
        exit(1);
    }

    h_values = convert_2D_to_1D_per_ragged_matrix(M, nz, nz_per_row, values);
    h_col_indices = convert_2D_to_1D_per_ragged_matrix(M, nz, nz_per_row, col_indices);

    printf("Allocating device variables for CPU ELLPACK product ...\n");

    err = hipMalloc((void **)&d_y, M * K * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device y (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_X, N * K * sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device X (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_values, M * nz * sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device values (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_col_indices, M * nz * sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device col_indices (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_nz_per_row, M * sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device nz_per_row (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_sum_nz, M * sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device sum_mz (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");

    err = hipMemcpy(d_values, h_values, M * nz * sizeof(double), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy values from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_col_indices, h_col_indices, M * nz * sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy ja from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_X, h_X, N * K * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy matrix X from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_nz_per_row, nz_per_row, M * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy nz_per_row from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    h_sum_nz = compute_sum_nz(M, nz_per_row);
    err = hipMemcpy(d_sum_nz, h_sum_nz, M * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy device sum_nz from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int numElements = M * K;
    int threadsPerBlock = 1024;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
           threadsPerBlock);

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // START TIMER
    checkCudaErrors(hipEventRecord(start, stream));

    ELLPACK_kernel<<<blocksPerGrid, threadsPerBlock>>>(M, K, d_nz_per_row, d_sum_nz, d_values, d_col_indices, d_X, d_y, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch ELLPACK kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // STOP TIMER
    checkCudaErrors(hipEventRecord(stop, stream));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&expireTimeMsec, start, stop));

    printf("ELAPSED TIME FOR PARALLEL PRODUCT GPU: %lf ns = %lf ms = %lf seconds\n", expireTimeMsec * 1e6, expireTimeMsec, expireTimeMsec * 1e-3);

    if (time != NULL)
        *time = expireTimeMsec * 1e6;
    printf("GFLOPS FOR PARALLEL PRODUCT GPU: %lf\n", compute_GFLOPS(K, nz, expireTimeMsec * 1e6));

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_y, d_y, M * K * sizeof(double), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_values);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device values(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_col_indices);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device col_indices(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_nz_per_row);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device nz_per_row(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_X);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix X (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_y);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix y (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_sum_nz);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device sum_nz (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    printf("Freeing host memory ...\n");
    if (h_X != NULL) free(h_X);
    if (h_values != NULL) free(h_values);
    if (h_col_indices != NULL) free(h_col_indices);
    if (h_sum_nz != NULL) free(h_sum_nz);

    printf("Completed parallel product ...\n");

    // for (int i = 0; i < M; i++)
    // {
    //     printf("\n");
    //     for (int z = 0; z < K; z++)
    //     {
    //         printf("y[%d][%d] = %.70lf\t", i, z, h_y[i*K + z]);
    //     }
    //     printf("\n");
    // }

    return h_y;
}