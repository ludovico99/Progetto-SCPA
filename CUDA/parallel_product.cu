#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <math.h>

#include <hip/hip_runtime.h> // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h> // For CUDA SDK timers
#include "../header.h"


__global__ void CSR_kernel(const int M, const int N, const int K, const int nz, double *d_as, int *d_ja, int *d_irp, double *d_X, double *d_y, int numElements)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    int i = tid / K;
    int z = tid % K;
    double partial_sum = 0;
    if (tid < numElements)
    {
        if (i == 0 && d_irp[i] == -1)
        {
            d_y[i*K + z]= 0.0;
        }
        if (i > 0 && d_irp[i] == d_irp[i - 1])
        {
             d_y[i*K + z] = 0.0;
        }
        else
        {   
            for (int j = d_irp[i]; (i < (M - 1) && j < d_irp[i + 1]) || (i >= M - 1 && j < nz); j++)
            {
                if (d_as != NULL) 
                    partial_sum += d_as[j] * d_X[d_ja[j]* K + z];
                else 
                    partial_sum += 1.0 *d_X [d_ja[j]*K + z];
            }
            d_y[i*K + z] = partial_sum;
        }
       
    }
}

double *convert_2D_to_1D (int M, int K, double **X){


    double * ret = (double*)malloc(M * K * sizeof(double));

    printf("Starting 2D conversion in 1D\n");
    for (int i = 0; i < M; i ++) {
        for (int j = 0; j < K; j++){
            ret[i*K + j] = X[i][j];
        }
        free(X[i]);
    }
    if (X != NULL) free(X);
    return ret;

}

double * CSR_GPU(int M, int N, int K, int nz, double *h_as, int *h_ja, int *h_irp, double **X, double * time)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    hipEvent_t start, stop;
    hipStream_t stream = NULL;

    double *h_y = NULL;
    double * h_X = NULL;
    double *d_y = NULL;

    double *d_X = NULL;

    double *d_as = NULL;
    int *d_ja = NULL;
    int *d_irp = NULL;

    float expireTimeMsec = 0.0;

    h_X = convert_2D_to_1D (M, K, X);

    h_y = (double *)malloc(M * K * sizeof(double));
    if (h_y == NULL)
    {
        printf("Errore malloc per y\n");
        exit(1);
    }

    // for (int i = 0; i < M; i++)
    // {
    //     h_y[i] = (double *)malloc(K * sizeof(double));
    //     if (h_y[i] == NULL)
    //     {
    //         printf("Errore malloc\n");
    //         exit(1);
    //     }
    // }
    printf("Allocating device variables for CPU CSR product ...\n");

    err = hipMalloc((void **)&d_y, M * K *sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device y (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_X, N * K *sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device irp (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_as, nz * sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device as (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_ja, nz * sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device ja (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_irp, M * sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device irp (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // err = hipMemset(d_as, 0.0, sizeof(double) * nz);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset as (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // err = hipMemset(d_irp, -1, sizeof(int) * M);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset irp (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // err = hipMemset(d_ja, 0, sizeof(int) * nz);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset ja (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // err = hipMemset2D(d_X, X_pitch, 0, X_width, X_height);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset ja (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // err = hipMemset2D(d_y, y_pitch, 0, y_width, y_height);
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr,
    //             "Failed to memset ja (error code %s)!\n",
    //             hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");

    err = hipMemcpy(d_as, h_as, nz * sizeof(double), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy as from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_ja, h_ja, nz * sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy ja from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_irp, h_irp, M * sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy irp from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_X, h_X, N * K * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy matrix X from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Launch the Vector Add CUDA Kernel
    int numElements = M * K;
    int threadsPerBlock = 1024;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
           threadsPerBlock);

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // START TIMER
    checkCudaErrors(hipEventRecord(start, stream));

    CSR_kernel<<<blocksPerGrid, threadsPerBlock>>>(M, N, K, nz, d_as, d_ja, d_irp, d_X, d_y, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch CSR kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // STOP TIMER
    checkCudaErrors(hipEventRecord(stop, stream));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&expireTimeMsec, start, stop));

    printf("ELAPSED TIME FOR PARALLEL PRODUCT GPU: %lf ns = %lf ms = %lf seconds\n",expireTimeMsec * 1e6, expireTimeMsec, expireTimeMsec*1e-3);
    
    //if (time != NULL) *time = expireTimeMsec * 1e6;
    printf("GFLOPS FOR PARALLEL PRODUCT GPU: %lf\n", compute_GFLOPS(K, nz, expireTimeMsec * 1e6));

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_y, d_y,M * K * sizeof(double), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_as);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device as(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_ja);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device ja(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_irp);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device irp(error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_X);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix X (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_y);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix y (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    // free(h_A);
    // free(h_B);
    // free(h_C);

    printf("Completed parallel product ...\n");

    // for (int i = 0; i < M; i++)
    // {
    //     printf("\n");
    //     for (int z = 0; z < K; z++)
    //     {
    //         printf("y[%d][%d] = %.70lf\t", i, z, h_y[i*K + z]);
    //     }
    //     printf("\n");
    // }

    return h_y;
}